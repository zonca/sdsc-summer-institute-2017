// SDSC Summer Institute 2017
// Andreas Goetz (agoetz@sdsc.edu)

// CUDA program to add two vectors in parallel on the GPU
// version 2:
// launch a fixed number of blocks and threads
//


#include <hip/hip_runtime.h>
#include<stdio.h>

// define vector length, number of blocks NBL and threads per block TPB
#define N (255*2047)
#define NBL 256
#define TPB 128

//
// CUDA device function that adds two integer vectors
//
__global__ void add(int *a, int *b, int *c, int n){

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = gridDim.x * blockDim.x;

  while (tid < n) {
    c[tid] = a[tid] + b[tid];
    tid += stride;
  }

}

//
// main program
//
int main(void){

  int h_a[N], h_b[N], h_c[N];
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);
  int i, err;

  // allocate device memory
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // initialize vectors
  for (i=0; i<N; i++){
    h_a[i] = i+1;
    h_b[i] = i+1;
  }

  // copy input data to device
  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

  // add vectors by launching a sufficient number of blocks of the add() kernel
  printf("\nLaunching vector addition kernel...\n");
  printf("Vector length     = %d\n",N);
  printf("Blocks            = %d\n",NBL);
  printf("Threads per block = %d\n",TPB);
  printf("Kernel copies     = %d\n",NBL*TPB);
  add<<<NBL,TPB>>>(d_a, d_b, d_c, N);

  // copy results back to host
  hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

  // deallocate memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // check results
  err = 0;
  for (i=0; i<N; i++){
    if (h_c[i] != 2*(i+1)) err = 1;
  }
  if (err != 0){
    printf("\n Error, %d elements do not match!\n\n", err);
  } else {
    printf("\n Success! All elements match.\n\n");
  }

  return 0;

}
